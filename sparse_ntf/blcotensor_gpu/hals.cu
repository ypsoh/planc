#include "hals.h"

void hals_update(MAT_GPU * fm, MAT_GPU * o_mttkrp_gpu, const MAT_GPU * gram) {
  _FType *fm_times_gram_col;
  _FType *mttkrp_t; // temp for transpose mttkrp

  int m = fm->n_rows;
  int n = fm->n_cols;

  // double * _check;
  // _check = (double *) malloc(sizeof(double) * n * n);
  // hipMemcpy(_check, gram->vals, sizeof(double) * n * n, hipMemcpyDeviceToHost);
  // for (int e = 0; e < n * n; ++e) {
  //   printf("%f ", _check[e]);
  //   if ((e+1)%n == 0) printf("\n");
  // }

  check_cuda(
    hipMalloc((void**)&fm_times_gram_col, m * sizeof(_FType)), "hipMalloc fm_times_gram_col"
  );

  int num_elements = m * n;
  int num_blocks = (num_elements + BLOCK_SIZE - 1 )/ BLOCK_SIZE;

  int rank = n;
  for (int r = 0; r < rank; ++r) {
    // get column from gram_without_one
    int col_idx = m * r;
    mat_vec_mul(fm->vals, &gram->vals[rank * r], fm_times_gram_col, m, n, 1.0, 0.0);

    num_elements = m;
    num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dvec_sub(fm->vals+col_idx, fm_times_gram_col, fm->vals+col_idx, m);
    dvec_add(fm->vals+col_idx, o_mttkrp_gpu->vals+col_idx, fm->vals+col_idx, m);
    __apply_threshold<<<num_blocks, BLOCK_SIZE>>>(fm->vals+col_idx, m, 1e-16, 1e-16);
  }

  hipFree(fm_times_gram_col);
  hipFree(mttkrp_t);
}